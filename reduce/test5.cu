#include "error.cuh"
#include <stdio.h>

int main(int argc, char *argv[])
{
	int device_id = 0;
	if (argc > 1) device_id = atoi(argv[1]);
	CHECK(hipSetDevice(device_id));
	hipDeviceProp_t prop;
	CHECK(hipGetDeviceProperties(&prop, device_id));
	printf("Device id:                 %d\n", device_id);
	printf("Device name:               %s\n", prop.name);
	printf("Compute capability:        %d.%d\n", prop.major, prop.minor);
	printf("Amount of global memory:   %g GB\n", prop.totalGlobalMem / (1024 * 1024 * 1024.0));
	printf("Amount of const memory:    %g KB\n", prop.totalConstMem / (1024.0));
	printf("Max grid size:             %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("Max block dim size:        %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Number of SMs:             %d\n", prop.multiProcessorCount);
	printf("Max amount of shared memory per block: %g KB\n", prop.sharedMemPerBlock/1024.0);
	printf("Max amount of shared memory per SM:    %g KB\n", prop.sharedMemPerMultiprocessor/1024.0);
	printf("Max number of regs per block:          %d K\n",    prop.regsPerBlock/1024);
	printf("Max number of regs per SM:             %d K\n",    prop.regsPerMultiprocessor/1024);
	printf("Max number of threads per block:       %d\n",    prop.maxThreadsPerBlock);
	printf("Max number of threads per SM:       %d = %d warps\n",    prop.maxThreadsPerMultiProcessor, prop.maxThreadsPerMultiProcessor/32);
	return 0;
}
